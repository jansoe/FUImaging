#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"matrix.h"
#include<time.h>
#include<sys/time.h>

// status printed and convergence check every ITER_CHECK iterations
#define ITER_CHECK 100
// max number of iterations
#define MAX_ITER 10000
// set to zero to guarantee MAX_ITER iterations, 0.001 is a good value otherwise
#define CONVERGE_THRESH 0.001

// number of timers used in profiling (don't change)
#define TIMERS 10  
char *tname[] = {"total","sgemm","eps","vecdiv","vecmult","sumrows","sumcols","coldiv","rowdiv","check"};


void update_div(matrix W, matrix H, matrix X, const float thresh, const int max_iter, double* t, int verbose);
double get_time();
unsigned nextpow2(unsigned x);


int main(int argc, char *argv[]){


    //factor X into W*H
    matrix W,H,X;

    // read in matrix data:
    // X - matrix to factorize
    // W - initial W matrix
    // H - initial H matrix
    read_matrix(&W,"../W.bin");
    read_matrix(&X,"../X.bin");
    read_matrix(&H,"../H.bin");

    //make sure no zero elements
    matrix_eps(X);
    matrix_eps(H);
    matrix_eps(W);

    int max_iter;
    if(argc > 1)
        max_iter = atoi(argv[1]);
    else 
        max_iter = MAX_ITER;

    // iterative nmf minimization
    update_div(W,H,X,CONVERGE_THRESH,max_iter,NULL,1);


    // write results matrices to binary files
    // (can be read with export_bin.m in Matlab)
    write_matrix(W,"../Wout.bin");
    write_matrix(H,"../Hout.bin");

    destroy_matrix(&W);
    destroy_matrix(&H);
    destroy_matrix(&X);
    return 0;

}

extern "C" {
int nmf(float *WP, float *HP, float *XP, int m, int n, int k){

    //factor X into W*H
    matrix W,H,X;

    // read in matrix data:
    // X - matrix to factorize
    // W - initial W matrix
    // H - initial H matrix
    read_matrix_from_float(&W, m, k, WP);
    read_matrix_from_float(&H, k, n, HP);
    read_matrix_from_float(&X, m, n, XP);

    //make sure no zero elements
    matrix_eps(X);
    matrix_eps(H);
    matrix_eps(W);


    // iterative nmf minimization
    update_div(W,H,X,CONVERGE_THRESH,MAX_ITER,NULL,0);

    return 0;
}
}


double get_time(){
    //output time in microseconds

    //the following line is required for function-wise timing to work,
    //but it slows down overall execution time.
    //comment out for faster execution
    hipDeviceSynchronize(); 

    struct timeval t;
    gettimeofday(&t,NULL);
    return (double)(t.tv_sec+t.tv_usec/1E6);
}

int start_time(double* t, int i)
{
    if (t != NULL)
    {
        t[i] -= get_time();
        return 1;
    }
    else 
        return 0;
}

int stop_time(double* t, int i)
{
    if (t != NULL)
    {
        t[i] += get_time();
        return 1;
    }
    else 
        return 0;
}



void update_div(matrix W0, matrix H0, matrix X0, const float thresh, const int max_iter, double *t,int verbose){
    //run iterative multiplicative updates on W,H

    cublasInit();

    const int M = W0.dim[0];
    const int K = W0.dim[1];
    const int N = H0.dim[1];

    // pad matrix dimensions to multiples of:
    const int PAD_MULT = 32;

    int M_padded = M;
    if (M%PAD_MULT != 0)
        M_padded = M + (PAD_MULT - (M % PAD_MULT));

    int K_padded = K;
    if (K%PAD_MULT != 0)
        K_padded = K + (PAD_MULT - (K % PAD_MULT));

    int N_padded = N;
    if (N%PAD_MULT != 0)
        N_padded = N + (PAD_MULT - (N % PAD_MULT));

    //unpadded test
    //M_padded = M;
    //N_padded = N;
    //K_padded = K;

    // find reduction parameters
    int MN_params[4] = {1,1,1,1}; //M*N size reduction (whole matrix)
    int N_params[4] = {1,1,1,1}; //N size reductions (rows)
    int M_params[4] = {1,1,1,1}; //M size reductions (cols)

    int rem;
    rem = nextpow2(N_padded/128 + (!(N_padded%128)?0:1));
    if (rem <= 128)
    {
        N_params[0] = 128;
        N_params[1] = rem;
    }
    else if (rem <= 512)
    {
        N_params[0] = rem;
        N_params[1] = 128;
    }
    else
    {
        fprintf(stderr,"reduction parameter error\n");
        exit(1);
    }


    rem = nextpow2(M_padded/128 + (!(M_padded%128)?0:1));
    if (rem <= 128)
    {
        M_params[0] = 128;
        M_params[1] = rem;
    }
    else if (rem <= 512)
    {
        M_params[0] = rem;
        M_params[1] = 128;
    }
    else
    {
        fprintf(stderr,"reduction parameter error\n");
        exit(1);
    }

    MN_params[0] = M_params[0];
    MN_params[1] = M_params[1];
    MN_params[2] = N_params[0];
    MN_params[3] = N_params[1];

    //printf("reduction parameters: ");
    //printf("%u,%u,%u,%u\n",MN_params[0],MN_params[1],MN_params[2],MN_params[3]);


    // block size in vector arithmetic operations
    const int BLOCK_SIZE = 1024;





    //copy host matrices to device memory
    copy_matrix_to_device(&W0);
    copy_matrix_to_device(&H0);
    copy_matrix_to_device(&X0);

    //matrix to hold W*H
    matrix WH0;
    create_matrix_on_device(&WH0,M,N,0.0);


    int i;

    /*
       double t_array[TIMERS];
       if(t==NULL)
       t = t_array;
       */
    if (t != NULL)
    {
        for(i=0;i<TIMERS;i++)
            t[i] = 0;
    }

    //float nancheck, zerocheck;
    // compute initial divergence and error
    float diff,div,change,prev_diff,prev_div;

    matrix_multiply_d(W0,H0,WH0);
    diff = matrix_difference_norm_d(compute,X0,WH0,MN_params);


    div = matrix_div_d(compute,X0,WH0,MN_params);
    if(verbose)
        printf("i: %4i, error: %6.4f, initial div: %8.4e\n",0,diff,div);


    // free device memory for unpadded matrices
    free_matrix_on_device(&W0);
    free_matrix_on_device(&H0);
    free_matrix_on_device(&X0);
    free_matrix_on_device(&WH0);


    //initialize temp matrices -----------------------


    //matrix to hold X./(W*H+EPS)
    matrix Z;
    create_matrix_on_device(&Z,M_padded,N_padded,0.0);

    //matrix to hold W'*Z
    matrix WtZ;
    create_matrix_on_device(&WtZ,K_padded,N_padded,0.0);

    //matrix to hold Z*H'
    matrix ZHt;
    create_matrix_on_device(&ZHt,M_padded,K_padded,0.0);

    //matrix to hold sum(W) [sum of cols of W]
    matrix sumW;
    create_matrix_on_device(&sumW,1,K_padded,0.0);

    //matrix to hold sum(H,2) [sum of rows of H]
    matrix sumH2;
    create_matrix_on_device(&sumH2,K_padded,1,0.0);


    //matrices to hold padded versions of matrices
    matrix W;
    create_matrix_on_device(&W,M_padded,K_padded,0.0);

    matrix H;
    create_matrix_on_device(&H,K_padded,N_padded,0.0);

    matrix X;
    create_matrix_on_device(&X,M_padded,N_padded,0.0);




    // move host matrices to padded device memory
    copy_matrix_to_device_padded(W0,W);
    copy_matrix_to_device_padded(H0,H);
    copy_matrix_to_device_padded(X0,X);




    //t[0] -= get_time();
    start_time(t,0);

        //matrix test1;

        for(i=0;i<max_iter;i++){

            //check for convergence, print status
            if(i % ITER_CHECK == 0 && i != 0){
                //t[9] -= get_time();
                start_time(t,9);
                matrix_multiply_d(W,H,Z);
                prev_diff = diff;
                diff = matrix_difference_norm_d(compute,X,Z,MN_params);
                change = (prev_diff-diff)/prev_diff;
                //t[9] += get_time();
                stop_time(t,9);
                if(verbose)
                    printf("i: %4i, error: %6.4f, %% change: %8.5f\n",
                            i,diff,change);
                if(change < thresh){
                    printf("converged\n");
                    break;
                }
            }


            /* matlab algorithm
               Z = X./(W*H+eps); H = H.*(W'*Z)./(repmat(sum(W)',1,F)); 
               Z = X./(W*H+eps);
               W = W.*(Z*H')./(repmat(sum(H,2)',N,1));
               */

            //
            // UPDATE H -----------------------------
            //


            //WH = W*H
            //t[1] -= get_time();
            start_time(t,1);
            matrix_multiply_d(W,H,Z);
            //t[1] += get_time();
            stop_time(t,1);




            //WH = WH+EPS
            //t[2] -= get_time();
            start_time(t,2);
            matrix_eps_d(Z,BLOCK_SIZE);
            //t[2] += get_time();
            stop_time(t,2);


            //Z = X./WH
            //t[3] -= get_time();
            start_time(t,3);
            element_divide_d(X,Z,Z,BLOCK_SIZE);
            //t[3] += get_time();
            stop_time(t,3);


            //sum cols of W into row vector
            //t[6] -= get_time();
            start_time(t,6);
            sum_cols_d(compute,W,sumW,M_params);
            matrix_eps_d(sumW,32);
            //t[6] += get_time();
            stop_time(t,6);

            //convert sumW to col vector (transpose)
            sumW.dim[0] = sumW.dim[1];
            sumW.dim[1] = 1;


            //WtZ = W'*Z
            //t[1] -= get_time();
            start_time(t,1);
            matrix_multiply_AtB_d(W,Z,WtZ);
            //t[1] += get_time();
            stop_time(t,1);


            //WtZ = WtZ./(repmat(sum(W)',1,H.dim[1])
            //[element divide cols of WtZ by sumW']
            //t[7] -= get_time();
            start_time(t,7);
            col_divide_d(WtZ,sumW,WtZ);
            //t[7] += get_time();
            stop_time(t,7);



            //H = H.*WtZ
            //t[4] -= get_time();
            start_time(t,4);
            element_multiply_d(H,WtZ,H,BLOCK_SIZE);
            //t[4] += get_time();
            stop_time(t,4);



            //
            // UPDATE W ---------------------------
            //

            //WH = W*H
            //t[1] -= get_time();
            start_time(t,1);
            matrix_multiply_d(W,H,Z);
            //t[1] += get_time();
            stop_time(t,1);


            //WH = WH+EPS
            //t[2] -= get_time();
            start_time(t,2);
            matrix_eps_d(Z,BLOCK_SIZE);
            //t[2] += get_time();
            stop_time(t,2);

            //Z = X./WH
            //t[3] -= get_time();
            start_time(t,3);
            element_divide_d(X,Z,Z,BLOCK_SIZE);
            //t[3] += get_time();
            stop_time(t,3);


            //sum rows of H into col vector
            //t[5] -= get_time();
            start_time(t,5);
            sum_rows_d(compute,H,sumH2,N_params);
            matrix_eps_d(sumH2,32);
            //t[5] += get_time();
            stop_time(t,5);

            //convert sumH2 to row vector (transpose)
            sumH2.dim[1] = sumH2.dim[0];
            sumH2.dim[0] = 1;

            //ZHt = Z*H'
            //t[1] -= get_time();
            start_time(t,1);
            matrix_multiply_ABt_d(Z,H,ZHt);
            //t[1] += get_time();
            stop_time(t,1);

            //ZHt = ZHt./(repmat(sum(H,2)',W.dim[0],1)
            //[element divide rows of ZHt by sumH2']
            //t[8] -= get_time();
            start_time(t,8);
            row_divide_d(ZHt,sumH2,ZHt);
            //t[8] += get_time();
            stop_time(t,8);

            //W = W.*ZHt
            //t[4] -= get_time();
            start_time(t,4);
            element_multiply_d(W,ZHt,W,BLOCK_SIZE);
            //t[4] += get_time();
            stop_time(t,4);


            // ------------------------------------

            //reset sumW to row vector
            sumW.dim[1] = sumW.dim[0];
            sumW.dim[0] = 1;
            //reset sumH2 to col vector
            sumH2.dim[0] = sumH2.dim[1];
            sumH2.dim[1] = 1;

            // ---------------------------------------

        }

    //t[0] += get_time();
    stop_time(t,0);




    //reallocate unpadded device memory
    allocate_matrix_on_device(&W0);
    allocate_matrix_on_device(&H0);

    //copy padded matrix to unpadded matrices
    copy_from_padded(W0,W);
    copy_from_padded(H0,H);

    // free padded matrices
    destroy_matrix(&W);
    destroy_matrix(&H);
    destroy_matrix(&X);

    // free temp matrices
    destroy_matrix(&Z);
    destroy_matrix(&WtZ);
    destroy_matrix(&ZHt);
    destroy_matrix(&sumW);
    destroy_matrix(&sumH2);

    copy_matrix_to_device(&X0);
    create_matrix_on_device(&WH0,M,N,0.0);

    // copy device results to host memory
    copy_matrix_from_device(&W0);
    copy_matrix_from_device(&H0);

    // evaluate final results
    matrix_multiply_d(W0,H0,WH0);
    prev_diff = diff;
    diff = matrix_difference_norm_d(compute,X0,WH0,MN_params);
    prev_div = div;
    div = matrix_div_d(compute,X0,WH0,MN_params);
    if(verbose){
        change = (prev_diff-diff)/prev_diff;
        printf("max iterations reached\n");
        printf("i: %4i, error: %6.4f, %% change: %8.5f\n",
                i,diff,change);
        change = (prev_div-div)/prev_div;
        printf("\tfinal div: %8.4e, %% div change: %8.5f\n",
                div,change);

        printf("\n");
        if (t != NULL)
        {
            for(i=0;i<TIMERS;i++)
                printf("t[%i]: %8.3f (%6.2f %%) %s\n",i,t[i],t[i]/t[0]*100,tname[i]);
        }
    }

    //clean up extra reduction memory
    matrix_difference_norm_d(cleanup,X0,WH0,MN_params);
    matrix_div_d(cleanup,X0,WH0,MN_params);
    sum_cols_d(cleanup,W,sumW,M_params);
    sum_rows_d(cleanup,H,sumH2,N_params);

    // free device memory for unpadded matrices
    free_matrix_on_device(&W0);
    free_matrix_on_device(&H0);
    free_matrix_on_device(&X0);

    // free temp matrices
    destroy_matrix(&WH0);

    cublasShutdown();

}

unsigned nextpow2(unsigned x) 
{
    x = x - 1;
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    return x + 1;

}
